#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void vectorMultiply(float *A, float *B, float *C, int numElements)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    if (i < numElements) C[i] = A[i] * B[i];
}

int main(void)
{

    // params
    hipError_t err = hipSuccess;
    float eps = 0.00001;
    int numElements = 50000;
    size_t size = numElements * sizeof(float);

    printf("[Vector addition of %d elements]\n", numElements);

    // allocate host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // initialize host vectors
    for (int i=0; i<numElements; i++)
    {
        h_A[i] = i + 1;
        h_B[i] = 1.0 / (i + 1 + eps);
    }

    // allocate device vectors
    float *d_A = NULL;
    float *d_B = NULL;
    float *d_C = NULL;
    hipError_t err1 = hipMalloc((void **)&d_A, size);
    hipError_t err2 = hipMalloc((void **)&d_B, size);
    hipError_t err3 = hipMalloc((void **)&d_C, size);
    if ((err1 != hipSuccess) || (err2 != hipSuccess) || (err3 != hipSuccess))
    {
        fprintf(stderr, "Failed to allocate one of the device vectors\n");
        exit(EXIT_FAILURE);
    }

    // copy host input vectors A and B
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(numElements / (float)threadsPerBlock);
    vectorMultiply <<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // copy device result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // verify that result is correct by computing on cpu
    for (int i = 0; i < numElements; i++)
    {
        if (fabs((h_A[i] * h_B[i]) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");

    // free device global memory and host memory
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);

    return 0;
}